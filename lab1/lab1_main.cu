#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <iomanip>
#include "csc.h"
#include "lab1.h"

constexpr int BLOCKS_NUM  = 128;
constexpr int THREADS_NUM = 256;

int main() {
    std::ios::sync_with_stdio(false);
    std::cin.tie(nullptr);

    int n;
    std::cin >> n;
    std::vector<double> h_a(n), h_b(n);
    for (size_t i = 0; i < n; ++i) {
        std::cin >> h_a[i];
    }
    for (size_t i = 0; i < n; ++i) {
        std::cin >> h_b[i];
    }


    double *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;
    size_t bytes = sizeof(double) * static_cast<size_t>(n);
    CSC(hipMalloc(&d_a, bytes));
    CSC(hipMalloc(&d_b, bytes));
    CSC(hipMalloc(&d_c, bytes));

    CSC(hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice));
    CSC(hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice));

    vecComponentWiseMultKernel<<<BLOCKS_NUM, THREADS_NUM>>>(d_a, d_b, d_c, n);
    CSC(hipGetLastError());
    CSC(hipDeviceSynchronize());

    std::vector<double> h_c(n);
    CSC(hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost));

    CSC(hipFree(d_a));
    CSC(hipFree(d_b));
    CSC(hipFree(d_c));

    std::cout << std::scientific << std::setprecision(10);
    for (int i = 0; i < n; ++i) {
        if (i) std::cout << ' ';
        std::cout << h_c[i];
    }
    std::cout << '\n';

    return 0;
}
