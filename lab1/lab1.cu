
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include <iomanip>

#define CSC(call)  									                \
do {											                    \
    hipError_t res = call;							                \
    if (res != hipSuccess) {							            \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
                __FILE__, __LINE__, hipGetErrorString(res));		\
        exit(0);								                    \
    }										                        \
} while(0)


__global__ void vecAddKernel(const double* a, const double* b, double* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

int main() {
    int n;
    std::cin >> n;
    std::vector<double> h_a(n), h_b(n);
    for (size_t i = 0; i < n; ++i) {
        std::cin >> h_a[i];
    }
    for (size_t i = 0; i < n; ++i) {
        std::cin >> h_b[i];
    }


    double *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;
    size_t bytes = sizeof(double) * (size_t)n;
    CSC(hipMalloc(&d_a, bytes));
    CSC(hipMalloc(&d_b, bytes));
    CSC(hipMalloc(&d_c, bytes));

    CSC(hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice));
    CSC(hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice));

    const int blockSize = 256;
    int gridSize = static_cast<int>((n + blockSize - 1) / blockSize);
    vecAddKernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    CSC(hipGetLastError());
    CSC(hipDeviceSynchronize());

    std::vector<double> h_c(n);
    CSC(hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost));

    CSC(hipFree(d_a));
    CSC(hipFree(d_b));
    CSC(hipFree(d_c));

    std::cout << std::scientific << std::setprecision(10);
    for (size_t i = 0; i < n; ++i) {
        std::cout << h_c[i] << ' ';
    }

    return 0;
}