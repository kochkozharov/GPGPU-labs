#include <iostream>
#include <hip/hip_runtime.h>
#include "lab1.h"
#include "csc.h"

int main() {
    const int N = 1 << 20;
    size_t bytes = N * sizeof(double);

    double* h_a = new double[N];
    double* h_b = new double[N];
    double* h_c = new double[N];

    for (int i = 0; i < N; ++i) {
        h_a[i] = static_cast<double>(i) * 0.001;
        h_b[i] = static_cast<double>(i) * 0.002;
    }

    double *d_a, *d_b, *d_c;
    CSC(hipMalloc(&d_a, bytes));
    CSC(hipMalloc(&d_b, bytes));
    CSC(hipMalloc(&d_c, bytes));

    CSC(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice));
    CSC(hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice));

    int blockSizes[] = {32, 64, 128, 256, 512, 1024};
    int gridSizes[] = {1, 2, 4, 8, 16, 32, 64, 128, 256, 512, 1024};

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    std::cout << "Benchmarking vecComponentWiseMultKernel (N = " << N << ")\n";
    std::cout << "Grid\tBlock\tTime(ms)\n";

    for (int g = 0; g < sizeof(gridSizes)/sizeof(gridSizes[0]); ++g) {
        for (int b = 0; b < sizeof(blockSizes)/sizeof(blockSizes[0]); ++b) {
            int grid = gridSizes[g];
            int block = blockSizes[b];

            if (block > 1024) continue; 

            hipEventRecord(start);
            vecComponentWiseMultKernel<<<grid, block>>>(d_a, d_b, d_c, N);
            hipEventRecord(stop);

            hipEventSynchronize(stop);
            float ms = 0;
            hipEventElapsedTime(&ms, start, stop);

            std::cout << grid << "\t" << block << "\t" << ms << "\n";

            hipDeviceSynchronize();
        }
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
